#include <stdio.h>
#include <hip/hip_runtime.h>


#define THREADS_PER_BLOCK 256
#define MULTIPLICATIONS 4096

/**
 * Multiply square matrix (n x n) by the vector of size n.
 * 
 * 
 * @param mat Input matrix.
 * @param vec Input vector.
 * @param out Output vector.
 * @param n   Dimension.  
 */
__global__ void matrix_vector_multiplication(float *mat, float *vec, float *out, int n) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        float sum = 0;
        
        for (int k = 0; k < MULTIPLICATIONS; k++)
            for (int i = 0; i < n; i++)
                sum += mat[i * n + tid] * vec[i];
        
        out[tid] = sum;            
    }
}

int main(int argc, char const *argv[]) 
{ 
    if (argc != 2) {
        fprintf(stderr, "usage: %s <n>\n", argv[0]);
        exit(1);
    }

    int n = atoi(argv[1]);

    if (n < 0) {
        fprintf(stderr, "Invalid size\n");
        exit(1);
    } 
    
    // Host & device inputs and outputs
    float *h_mat, *h_vec, *h_out;
    float *d_mat, *d_vec, *d_out;

    // Allocate host memory
    h_mat = (float*) malloc(sizeof(float) * n * n);
    h_vec = (float*) malloc(sizeof(float) * n);
    h_out = (float*) malloc(sizeof(float) * n);

    // Allocate device memory 
    hipMalloc((void**) &d_mat, sizeof(float) * n * n);
    hipMalloc((void**) &d_vec, sizeof(float) * n);
    hipMalloc((void**) &d_out, sizeof(float) * n);

    // Initialize host matrix
    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++)
            h_mat[i * n + j] = 0.05;
    
    // Initialize host vector
    for (int i = 0; i < n; i++)
        h_vec[i] = 0.05;

	// Transfer data from host to device memory
	hipMemcpy(d_mat, h_mat, sizeof(float) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(d_vec, h_vec, sizeof(float) * n, hipMemcpyHostToDevice);
    
    hipEvent_t start, finish;
    float elapsed_milliseconds;

    // Use event to calculate time
	hipEventCreate(&start); 
	hipEventCreate(&finish);

	hipEventRecord(start, 0);
    matrix_vector_multiplication<<<n/THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK>>>(d_mat, d_vec, d_out, n);
    hipEventRecord(finish, 0); 

	hipEventSynchronize(finish);  // Wait for stop event to complete
	hipEventElapsedTime(&elapsed_milliseconds, start, finish);  // Calculate the time difference (millisecond level)

    // Transfer data back to host memory
	hipMemcpy(h_out, d_out, sizeof(float) * n, hipMemcpyDeviceToHost);

    printf("%f\n", h_out[0]);
    printf("GPU Elapsed time = %.2fs\n", elapsed_milliseconds/1000.0);
    
    // Deallocate device memory
    hipFree(d_mat);
    hipFree(d_vec);
    hipFree(d_vec);

    // Deallocate host memory
    free(h_mat); 
    free(h_vec); 
    free(h_out);

    return 0;
}